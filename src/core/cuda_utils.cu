#include "hip/hip_runtime.h"
#include "cuda_utils.h"
#include "cuda_defs.h"

#include "thrust/device_vector.h"
#include "thrust/device_ptr.h"
#include "thrust/transform.h"
#include "thrust/copy.h"
#include "thrust/gather.h"
#include "thrust/count.h"


// ---------------------------------------------------------------------
/*
	Functors 
*/ 
// ---------------------------------------------------------------------
template <typename T>
struct constant_add
{
	constant_add(T _val)
		: val(_val)
	{}

	__host__ __device__ 
	T operator() (T x) 
	{
		return x + val; 
	}

	T val;
};

template <typename T>
struct constant_sub
{
	constant_sub(T _val)
		: val(_val)
	{}

	__host__ __device__ 
	T operator() (T x) 
	{
		return x - val; 
	}

	T val;
};

template <typename T>
struct constant_mul
{
	constant_mul(T _val)
		: val(_val)
	{}

	__host__ __device__ 
	T operator() (T x) 
	{
		return x * val; 
	}

	T val;
};

struct is_valid
{
	__host__ __device__
		bool operator() (uint32 x)
	{
		return (x == 1);
	}
};


// ---------------------------------------------------------------------
/*
	Kernels
*/ 
// ---------------------------------------------------------------------

template <typename V, typename S>
__global__ void kernel_scale_vector_array(V *d_vec, uint32 count, S scalar)
{
	uint32 tid = threadIdx.x + blockIdx.x * blockDim.x;
	
	if (tid < count)
		d_vec[tid] *= scalar; 
}


// ---------------------------------------------------------------------
/*
/// \brief	Moves data from device memory \a d_vals to device memory \a d_array using \em source
/// 		addresses specified in \a d_srcAddr.
/// 		
/// 		\code d_array[i] = d_vals[d_srcAddr[i]] \endcode
/// 		
/// 		When the source address is \c 0xffffffff, the corresponding target entry will get zero'd.
///			This can be helpful for some algorithms.
/// 		
/// 		\warning	Heavy uncoalesced access possible. Depends on addresses. 
*/ 
// ---------------------------------------------------------------------
template <class T>
__global__ void kernel_set_from_address(T* d_array, uint* d_src_addr, T* d_vals, uint count_target)
{
	uint idx = blockIdx.x * blockDim.x + threadIdx.x;

	if(idx < count_target)
	{
		uint addr = d_src_addr[idx];
		T val = {0};
		if(addr != 0xffffffff)
			val = d_vals[addr];
		d_array[idx] = val;
	}
}

__global__ void kernel_init_identity(uint32 *d_buffer, uint32 count)
{
	uint32 idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < count)
		d_buffer[idx] = idx;
}


// ---------------------------------------------------------------------
/*
	Kernel Wrappers
*/ 
// ---------------------------------------------------------------------
template void kernel_wrapper_scale_vector_array<float4, float>(float4 *d_vec, uint32 count, float scalar); 


extern "C++"
template <typename V, typename S> 
void kernel_wrapper_scale_vector_array(V *d_vec, uint32 count, S scalar)
{
	dim3 block_size = dim3(256, 1, 1); 
	dim3 grid_size = dim3(CUDA_DIVUP(count, block_size.x), 1, 1); 
	
	kernel_scale_vector_array<V, S><<<grid_size, block_size>>>(d_vec, count, scalar);
}

extern "C++"
template <typename T>
void kernel_wrapper_set_from_address(T *d_array, uint32 *d_src_addr, T *d_vals, uint32 count_target)
{
	dim3 block_size = dim3(256, 1, 1);
	dim3 grid_size = dim3(CUDA_DIVUP(count_target, block_size.x), 1, 1);
	
	kernel_set_from_address<<<grid_size, block_size>>>(d_array, d_src_addr, d_vals, count_target);
}

extern "C++"
template <typename T> 
void kernel_wrapper_reduce(T& result, T *d_data, uint32 count, T identity)
{
	/*
	uint32 num_blocks = CUDA_DIVUP(count, 256);
	
	hipError_t err;
	T *d_block_res; 
	
	c_cuda_mem_pool& mem_pool = c_cuda_mem_pool::get_instance();
	err = mem_pool.request((void**)&d_block_res, num_blocks*sizeof(T), "temp", 64*sizeof(T)/4);
	if (err != hipSuccess)
		return err; 
	*/
}

extern "C"
void kernel_wrapper_init_identity(uint32 *d_buffer, uint32 count)
{
	dim3 block_size = dim3(256, 1, 1);
	dim3 grid_size = dim3(CUDA_DIVUP(count, block_size.x), 1, 1);
	
	kernel_init_identity<<<grid_size, block_size>>>(d_buffer, count);
	CUDA_CHECKERROR;
}

extern "C++"
template <typename T> 
void device_constant_add(T *d_array, uint32 count, T constant)
{
	thrust::device_ptr<T> d_ptr = thrust::device_pointer_cast(d_array); 
	thrust::transform(d_ptr, d_ptr+count, d_ptr, constant_add<T>(constant));
}
extern "C++"
template <typename T> 
void device_constant_sub(T *d_array, uint32 count, T constant)
{
	thrust::device_ptr<T> d_ptr = thrust::device_pointer_cast(d_array); 
	thrust::transform(d_ptr, d_ptr+count, d_ptr, constant_sub<T>(constant));
}
extern "C++"
template <typename T> 
void device_constant_mul(T *d_array, uint32 count, T constant)
{
	thrust::device_ptr<T> d_ptr = thrust::device_pointer_cast(d_array); 
	thrust::transform(d_ptr, d_ptr+count, d_ptr, constant_mul<T>(constant));
}

extern "C++"
template <typename T> 
void device_compact(T *d_in, unsigned *d_stencil, size_t num_elems, T *d_out_campacted, uint32 *d_out_new_count)
{	
	thrust::device_ptr<unsigned> d_stencil_ptr = thrust::device_pointer_cast(d_stencil);
	thrust::device_ptr<T> d_in_ptr = thrust::device_pointer_cast(d_in); 
	thrust::device_ptr<T> d_out_ptr = thrust::device_pointer_cast(d_out_campacted);

	uint32 new_count = thrust::copy_if(d_in_ptr, d_in_ptr+num_elems, d_stencil_ptr, d_out_ptr, is_valid()) - d_out_ptr;

	cuda_safe_call_no_sync(hipMemcpy(d_out_new_count, &new_count, sizeof(uint32), hipMemcpyHostToDevice)); 
}

template void device_constant_add<float>(float *d_array, uint32 count, float constant); 
template void device_constant_sub<float>(float *d_array, uint32 count, float constant); 
template void device_constant_mul<float>(float *d_array, uint32 count, float constant); 
template void device_constant_add<uint32>(uint32 *d_array, uint32 count, uint32 constant); 
template void device_constant_sub<uint32>(uint32 *d_array, uint32 count, uint32 constant); 
template void device_constant_mul<uint32>(uint32 *d_array, uint32 count, uint32 constant); 

template void device_compact<uint32>(uint32 *d_in, unsigned *d_stencil, size_t num_elems, uint32 *d_out_campacted, uint32 *d_out_new_count);


template void kernel_wrapper_set_from_address<uint32>(uint32 *d_array, uint32 *d_src_addr, uint32 *d_vals, uint32 count_target);
template void kernel_wrapper_set_from_address<float>(float *d_array, uint32 *d_src_addr, float *d_vals, uint32 count_target);
template void kernel_wrapper_set_from_address<float2>(float2 *d_array, uint32 *d_src_addr, float2 *d_vals, uint32 count_target);
template void kernel_wrapper_set_from_address<float4>(float4 *d_array, uint32 *d_src_addr, float4 *d_vals, uint32 count_target);